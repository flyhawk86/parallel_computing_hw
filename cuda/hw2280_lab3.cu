#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RANGE 11.79

/*** TODO: insert the declaration of the kernel function below this line ***/

__global__ 
void vecGPU(float* ab, float* bd, float* cd, int n);


/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	start = clock();
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
		3. write the kernel, call it: vecGPU
		4. Call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to write the number of threads, blocks, etc and their geometry.
		5. Bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	

	// 1. allocate ad, bd, and cd in the device
	hipMalloc((void**)&ad, n*sizeof(float));
	hipMalloc((void**)&bd, n*sizeof(float));
	hipMalloc((void**)&cd, n*sizeof(float));

	// 2. send a, b, and c to the device
	hipMemcpy(ad, a, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bd, b, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cd, c, n*sizeof(float), hipMemcpyHostToDevice);

	/* 
		4. Call the kernel
		1d block with 500 threads, 1d grid of 4 blocks
	*/

  	vecGPU<<<4,500>>>(ad, bd, cd, n);

  	/* 
  		5. copy the cd array in the device 
  		back to the c array inthe host
  	*/
  	hipMemcpy(c, cd, n*sizeof(float), hipMemcpyDeviceToHost);

  	// 6. free ad, bd, and cd
  	hipFree(ad);
  	hipFree(bd);
  	hipFree(cd);
	

	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	  if( fabsf(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		printf("Element %d in the result array does not match the sequential version\n", i);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/

__global__ 
void vecGPU(float* ad, float* bd, float* cd, int n){
	// the unique id of the current thread
	int thread_id = threadIdx.x + blockIdx.x*blockDim.x;

	// increment thread_id until it's bigger than n
	for(; thread_id < n; thread_id += gridDim.x*blockDim.x)
		cd[thread_id] += ad[thread_id] * bd[thread_id];
}